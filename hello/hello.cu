#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hello.hpp"

__global__ void hello_kernel(int value) {
    printf("Hello from kernel! %d\n", value);
}

void execute_hello_kernel(int value) {
    printf("Hello from host! %d\n", value);
    hello_kernel<<<1,1>>>(value);
    // wait for the kernel to finish
    hipDeviceSynchronize();
}
